#include "hip/hip_runtime.h"
#include <iostream>

#include "DeviceGrid.hpp"
#include "CollOfIndices.hpp"


#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <opm/common/ErrorMacros.hpp>

using namespace equelleCUDA;


void test_copy(CollOfFace coll) {
    if ( coll.size() != 10 ) {
	OPM_THROW(std::runtime_error, "\ntest_copy - size don't match");
    }
    if ( coll.isFull() ) {
	OPM_THROW(std::runtime_error, "\ntest_copy - collection says it is " << coll.isFull() << " but should be false.");
    }
    for (thrust::device_vector<int>::iterator it = coll.begin(); it != coll.end(); it++) {
	//for (int i = 0; i < coll.size(); i++) {
	if ( *it != 2 ){
	    OPM_THROW(std::runtime_error, "\ntest_copy - Collection don't match. Expected 2, got " << *it);
	}
    }
}

void test_full(CollOfCell coll, int s) {
    if ( !coll.isFull() ) {
	OPM_THROW(std::runtime_error, "\ntest_full - isFull() should be true but is " << coll.isFull());
    }
    if ( coll.size() != s ) {
	OPM_THROW(std::runtime_error, "\ntest_full - vector should have size " << s << ", but has size " << coll.size());
    }

}


__global__ void addOne(int* array, int size) {
    if ( blockIdx.x < size ) {
	array[blockIdx.x]++;
    }
}

void test_back_to_host(CollOfCell coll) {
    if (coll.size() != 20) {
	OPM_THROW(std::runtime_error, "\ntest_back_to_host - size should be 20, is " << coll.size());
    }
    thrust::host_vector<int> back = coll.toHost();
    for(int i = 0; i < back.size(); i++) {
	if (back[i] != i) {
	    OPM_THROW(std::runtime_error, "\ntest_back_to_host - back[" << i << "] should be " << i << " but is " << back[i]);
	}
    }
    
    // Get the raw pointer and call a kernel to add 1 on each
    int* ptr = 0; 
    ptr = coll.raw_pointer();
    if (ptr == 0) {
	OPM_THROW(std::runtime_error, "\ntest_back_to_host - failed assigning raw pointer");
    } 
    addOne<<<coll.size(), 1>>>(ptr, coll.size());
    // copy back to host again
    thrust::host_vector<int> added = coll.toHost();
    for(int i = 0; i < added.size(); ++i) {
	if ( added[i] != i+1 ) {
	    OPM_THROW(std::runtime_error, "\ntest_back_to_host - added[" << i << "] should be " << i+1 << " but is " << back[i]);
	}
    }

}

int cuda_main() {

    thrust::host_vector<int> host(10, 2);
    CollOfFace coll(host);

    CollOfCell cells(host);

    test_copy(coll);
    
    CollOfFace coll2 = coll;
    test_copy(coll2);

    int dummy_size = 20;
    CollOfCell coll3(dummy_size);
    test_full(coll3, dummy_size);

    thrust::host_vector<int> host2(0);
    for( int i = 0; i < 20; i++) {
	host2.push_back(i);
    }
    CollOfCell coll4(host2);
    test_back_to_host(coll4);

    return 0;
}
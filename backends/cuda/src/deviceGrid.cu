#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

//#include "EquelleRuntimeCUDA.hpp"

#include <iostream>
#include <string>
#include <vector>

// For error exception macro:
#include <opm/common/ErrorMacros.hpp>
#include <opm/grid/GridManager.hpp>
#include <opm/grid/UnstructuredGrid.h>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <thrust/memory.h>
#include <thrust/fill.h>
#include <thrust/detail/raw_pointer_cast.h>
#include <thrust/remove.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/retag.h>

#include "DeviceGrid.hpp"
#include "wrapDeviceGrid.hpp"
#include "CollOfScalar.hpp"
#include "CollOfIndices.hpp"
#include "CollOfVector.hpp"
#include "equelleTypedefs.hpp"
#include "device_functions.cuh"


using namespace equelleCUDA;
using namespace wrapDeviceGrid;


// --------------------------------------------------- //
// -------- Implementation of DeviceGrid ------------- //
// --------------------------------------------------- //


// Default constructor
DeviceGrid::DeviceGrid()
    : dimensions_(0),
      number_of_cells_(0),
      number_of_faces_(0),
      size_cell_faces_(0),
      cell_centroids_(0),
      face_centroids_(0),
      cell_facepos_(0),
      cell_faces_(0),
      cell_volumes_(0),
      face_areas_(0),
      face_cells_(0),
      face_normals_(0),
      boundary_faces_(),
      interior_faces_(),
      boundary_cells_(),
      interior_cells_(),
      boundaryFacesEmpty_(true),
      interiorFacesEmpty_(true),
      boundaryCellsEmpty_(true),
      interiorCellsEmpty_(true)
{
    // intentionally left blank
}

// Constructor from a OPM UnstructuredGrid struct
DeviceGrid::DeviceGrid( const UnstructuredGrid& grid)
    : dimensions_(grid.dimensions),
      number_of_cells_(grid.number_of_cells),
      number_of_faces_(grid.number_of_faces),
      size_cell_faces_(grid.cell_facepos[number_of_cells_]),
      cell_centroids_(0),
      face_centroids_(0),
      cell_facepos_(0),
      cell_faces_(0),
      cell_volumes_(0),
      face_areas_(0),
      face_cells_(0),
      face_normals_(0),
      boundary_faces_(),
      interior_faces_(),
      boundary_cells_(),
      interior_cells_(),
      boundaryFacesEmpty_(true),
      interiorFacesEmpty_(true),
      boundaryCellsEmpty_(true),
      interiorCellsEmpty_(true)
{
    // Allocate memory for cell_centroids_:
    // type: double
    // size: dimensions_ * number_of_cells_
    cudaStatus_ = hipMalloc( (void**)&cell_centroids_ ,
			      dimensions_ * number_of_cells_ * sizeof(double));
    checkError_("hipMalloc(cell_centroids_) in DeviceGrid::DeviceGrid(UnstructuredGrid&)");
    cudaStatus_ = hipMemcpy( cell_centroids_, grid.cell_centroids,
			      dimensions_ * number_of_cells_ * sizeof(double),
			      hipMemcpyHostToDevice );
    checkError_("hipMemcpy(cell_centroids_) in DeviceGrid::DeviceGrid(UnstructuredGrid&)");

    // Allocate memory for face_centroids_:
    // type: double
    // size: dimensions_ * number_of_faces_
    cudaStatus_ = hipMalloc( (void**)&face_centroids_ ,
			      dimensions_ * number_of_faces_ * sizeof(double));
    checkError_("hipMalloc(face_centroids_) in DeviceGrid::DeviceGrid(UnstructuredGrid&)");
    cudaStatus_ = hipMemcpy( face_centroids_, grid.face_centroids,
			      dimensions_ * number_of_faces_ * sizeof(double),
			      hipMemcpyHostToDevice );
    checkError_("hipMemcpy(face_centroids_) in DeviceGrid::DeviceGrid(UnstructuredGrid&)");

    // Allocate memory for cell_facepos_:
    // type: int
    // size: number_of_cells_ + 1
    cudaStatus_ = hipMalloc( (void**)&cell_facepos_, 
			      (number_of_cells_ + 1) * sizeof(int));
    checkError_("hipMalloc(cell_facepos_) in DeviceGrid::DeviceGrid(UnstructuredGrid&)");
    cudaStatus_ = hipMemcpy( cell_facepos_, grid.cell_facepos,
			      (number_of_cells_ + 1) * sizeof(int),
			      hipMemcpyHostToDevice );
    checkError_("hipMemcpy(cell_facepos_) in DeviceGrid::DeviceGrid(UnstructuredGrid&)");
    
    // Allocate memory for cell_faces:
    // type: int
    // size: cell_facepos_[ number_of_cells_ ]
    cudaStatus_ = hipMalloc( (void**)&cell_faces_, 
			      size_cell_faces_ * sizeof(int));
    checkError_("hipMalloc(cell_faces_) in DeviceGrid::DeviceGrid(UnstructuredGrid&)");
    cudaStatus_ = hipMemcpy( cell_faces_, grid.cell_faces,
			      size_cell_faces_ * sizeof(int),
			      hipMemcpyHostToDevice );
    checkError_("hipMemcpy(cell_faces_) in DeviceGrid::DeviceGrid(UnstructuredGrid&)");

    // Allocate memory for cell_volumes_:
    // type: double
    // size: number_of_cells_
    cudaStatus_ = hipMalloc( (void**)&cell_volumes_, 
			      number_of_cells_ * sizeof(double));
    checkError_("hipMalloc(cell_volumes_) in DeviceGrid::DeviceGrid(UnstructuredGrid&)");
    cudaStatus_ = hipMemcpy( cell_volumes_, grid.cell_volumes,
			      number_of_cells_ * sizeof(double),
			      hipMemcpyHostToDevice );
    checkError_("hipMemcpy(cell_volumes_) in DeviceGrid::DeviceGrid(UnstructuredGrid&)");

    // Allocate memory for face_areas_:
    // type: double
    // size: number_of_faces_
    cudaStatus_ = hipMalloc( (void**)&face_areas_, 
			      number_of_faces_ * sizeof(double));
    checkError_("hipMalloc(face_areas_) in DeviceGrid::DeviceGrid(UnstructuredGrid&)");
    cudaStatus_ = hipMemcpy( face_areas_, grid.face_areas,
			      number_of_faces_ * sizeof(double),
			      hipMemcpyHostToDevice );
    checkError_("hipMemcpy(face_areas_) in DeviceGrid::DeviceGrid(UnstructuredGrid&)");
 
    // Allocate memory for face_cells_:
    // type: int
    // size: 2 * number_of_faces_
    cudaStatus_ = hipMalloc( (void**)&face_cells_, 
			      2 * number_of_faces_ * sizeof(int));
    checkError_("hipMalloc(face_cells_) in DeviceGrid::DeviceGrid(UnstructuredGrid&)");
    cudaStatus_ = hipMemcpy( face_cells_, grid.face_cells,
			      2 * number_of_faces_ * sizeof(int),
			      hipMemcpyHostToDevice );
    checkError_("hipMemcpy(face_cells_) in DeviceGrid::DeviceGrid(UnstructuredGrid&)");

    // Allocate memory for face_normals_:
    // type: double
    // size: dimensions_ * number_of_faces_
    cudaStatus_ = hipMalloc( (void**)&face_normals_, 
			      dimensions_ * number_of_faces_ * sizeof(double));
    checkError_("hipMalloc(face_normals_) in DeviceGrid::DeviceGrid(UnstructuredGrid&)");
    cudaStatus_ = hipMemcpy( face_normals_, grid.face_normals,
			      dimensions_ * number_of_faces_ * sizeof(double),
			      hipMemcpyHostToDevice );
    checkError_("hipMemcpy(face_normals_) in DeviceGrid::DeviceGrid(UnstructuredGrid&)");

    // Normalize the face normals. They are scaled relative to the area of the faces.
    kernelSetup s(number_of_faces_);
    normalizeAllFaceNormals<<<s.grid,s.block>>>(face_normals_, face_areas_, number_of_faces_, dimensions_);
    hipDeviceSynchronize();

} // Constructor from OPMs UnstructuredGrid


// Copy constructor:
DeviceGrid::DeviceGrid(const DeviceGrid& grid) 
  : dimensions_(grid.dimensions_),
    number_of_cells_(grid.number_of_cells_),
    number_of_faces_(grid.number_of_faces_),
    size_cell_faces_(grid.size_cell_faces_),
    cell_centroids_(0),
    face_centroids_(0),
    cell_facepos_(0),
    cell_faces_(0),
    cell_volumes_(0),
    face_areas_(0),
    face_cells_(0),
    face_normals_(0),
    boundary_faces_(),
    interior_faces_(),
    boundary_cells_(),
    interior_cells_(),
    boundaryFacesEmpty_(true),
    interiorFacesEmpty_(true),
    boundaryCellsEmpty_(true),
    interiorCellsEmpty_(true)
{    
    // CELL_CENTROIDS_
    cudaStatus_ = hipMalloc( (void**)&cell_centroids_,
			      dimensions_ * number_of_cells_ * sizeof(double));
    checkError_("hipMalloc(cell_centroids_) in DeviceGrid::DeviceGrid(const DeviceGrid&)");
    cudaStatus_ = hipMemcpy( cell_centroids_, grid.cell_centroids_,
			      dimensions_ * number_of_cells_ * sizeof(double),
			      hipMemcpyDeviceToDevice );
    checkError_("hipMemcpy(cell_centroids_) in DeviceGrid::DeviceGrid(const DeviceGrid&)");
    // FACE_CENTROIDS_
    cudaStatus_ = hipMalloc( (void**)&face_centroids_,
			      dimensions_ * number_of_faces_ * sizeof(double));
    checkError_("hipMalloc(face_centroids_) in DeviceGrid::DeviceGrid(const DeviceGrid&)");
    cudaStatus_ = hipMemcpy( face_centroids_, grid.face_centroids_,
			      dimensions_ * number_of_faces_ * sizeof(double),
			      hipMemcpyDeviceToDevice);
    checkError_("hipMemcpy(face_centroids_) in DeviceGrid::DeviceGrid(const DeviceGrid&)");

    // CELL_FACEPOS_
    cudaStatus_ = hipMalloc( (void**)&cell_facepos_, 
			      (number_of_cells_ + 1) * sizeof(int));
    checkError_("hipMalloc(cell_facepos_) in DeviceGrid::DeviceGrid(const DeviceGrid&)");
    cudaStatus_ = hipMemcpy( cell_facepos_, grid.cell_facepos_,
			      (number_of_cells_ + 1) * sizeof(int),
			      hipMemcpyDeviceToDevice );
    checkError_("hipMemcpy(cell_facepos_) in DeviceGrid::DeviceGrid(const DeviceGrid&)");

    // CELL_FACES_
    cudaStatus_ = hipMalloc( (void**)&cell_faces_,
			      size_cell_faces_ * sizeof(int));
    checkError_("hipMalloc(cell_faces_) in DeviceGrid::DeviceGrid(const DeviceGrid&)");
    cudaStatus_ = hipMemcpy( cell_faces_, grid.cell_faces_,
			      size_cell_faces_ * sizeof(int),
			      hipMemcpyDeviceToDevice );
    checkError_("hipMemcpy(cell_faces_) in DeviceGrid::DeviceGrid(const DeviceGrid&)");

    // CELL_VOLUMES_
    cudaStatus_ = hipMalloc( (void**)&cell_volumes_,
			      number_of_cells_ * sizeof(double));
    checkError_("hipMalloc(cell_volumes_) in DeviceGrid::DeviceGrid(const DeviceGrid&)");
    cudaStatus_ = hipMemcpy( cell_volumes_, grid.cell_volumes_,
			      number_of_cells_ * sizeof(double),
			      hipMemcpyDeviceToDevice);
    checkError_("hipMemcpy(cell_volumes_) in DeviceGrid::DeviceGrid(const DeviceGrid&)");

    // FACE_AREAS_
    cudaStatus_ = hipMalloc( (void**)&face_areas_,
			      number_of_faces_ * sizeof(double));
    checkError_("hipMalloc(face_areas_) in DeviceGrid::DeviceGrid(const DeviceGrid&)");
    cudaStatus_ = hipMemcpy( face_areas_, grid.face_areas_,
			      number_of_cells_ * sizeof(double),
			      hipMemcpyDeviceToDevice);
    checkError_("hipMemcpy(face_areas_) in DeviceGrid::DeviceGrid(const DeviceGrid&)");
    
    // FACE_CELLS_
    cudaStatus_ = hipMalloc( (void**)&face_cells_,
			      2 * number_of_faces_ * sizeof(int));
    checkError_("hipMalloc(face_cells_) in DeviceGrid::DeviceGrid(const DeviceGrid&)");
    cudaStatus_ = hipMemcpy( face_cells_, grid.face_cells_,
			      2 * number_of_faces_ * sizeof(int),
			      hipMemcpyDeviceToDevice);
    checkError_("hipMemcpy(face_cells_) in DeviceGrid::DeviceGrid(const DeviceGrid&)");
			    
    // FACE_NORMALS
    cudaStatus_ = hipMalloc( (void**)&face_normals_,
			      dimensions_ * number_of_faces_ * sizeof(double));
    checkError_("hipMalloc(face_normals_) in DeviceGrid::DeviceGrid(const DeviceGrid&)");
    cudaStatus_ = hipMemcpy( face_normals_, grid.face_normals_,
			      dimensions_ * number_of_faces_ * sizeof(double),
			      hipMemcpyDeviceToDevice);
    checkError_("hipMemcpy(face_normals_) in DeviceGrid::DeviceGrid(const DeviceGrid&)");

    // Normalize the face normals. They are scaled relative to the area of the faces.
    kernelSetup s(number_of_faces_);
    normalizeAllFaceNormals<<<s.grid,s.block>>>(face_normals_, face_areas_, number_of_faces_, dimensions_);
    hipDeviceSynchronize();
} // copy constructor


// Destructor
DeviceGrid::~DeviceGrid() {

    if( cell_centroids_ != 0 ) {
	cudaStatus_ = hipFree(cell_centroids_);
	checkError_("hipFree(cell_centroids_) in DeviceGrid::~DeviceGrid()");
    }
    if ( face_centroids_ != 0 ) {
	cudaStatus_ = hipFree(face_centroids_);
	checkError_("hipFree(face_centroids) in DeviceGrid::~DeviceGrid()");
    }
    if ( cell_facepos_ != 0 ) {
	cudaStatus_ = hipFree(cell_facepos_);
	checkError_("hipFree(cell_facepos_) in DeviceGrid::~DeviceGrid()");
    }
    if ( cell_faces_ != 0 ) {
	cudaStatus_ = hipFree(cell_faces_);
	checkError_("hipFree(cell_faces_) in DeviceGrid::~DeviceGrid()");
    }
    if ( cell_volumes_ != 0 ) {
	cudaStatus_ = hipFree(cell_volumes_);
	checkError_("hipFree(cell_volumes_) in DeviceGrid::~DeviceGrid()");
    }
    if ( face_areas_ != 0 ) {
	cudaStatus_ = hipFree(face_areas_);
	checkError_("hipFree(face_areas_) in DeviceGrid::~DeviceGrid()");
    }
    if ( face_cells_ != 0 ) {
	cudaStatus_ = hipFree(face_cells_);
	checkError_("hipFree(face_cells_) in DeviceGrid::~DeviceGrid()");
    }
    if ( face_normals_ != 0 ) {
	cudaStatus_ = hipFree(face_normals_);
	checkError_("hipFree(face_normals_) in DeviceGrid::~DeviceGrid()");
    }

} // Destructor



// ------------ GRID OPERATIONS! ------------
CollOfCell DeviceGrid::allCells() const {
    return CollOfCell(number_of_cells_);
}

CollOfFace DeviceGrid::allFaces() const {
    return CollOfFace(number_of_faces_);
}


const CollOfFace& DeviceGrid::boundaryFaces() const {
    if ( boundaryFacesEmpty_ ) {
	createBoundaryFaces_();
    }
    return boundary_faces_;
}

const CollOfFace& DeviceGrid::interiorFaces() const {
    if ( interiorFacesEmpty_ ) {
	createInteriorFaces_();
    }
    return interior_faces_;
}

const CollOfCell& DeviceGrid::boundaryCells() const {
    if ( boundaryCellsEmpty_ ) {
	createBoundaryCells_();
    }
    return boundary_cells_;
}

const CollOfCell& DeviceGrid::interiorCells() const {
    if ( interiorCellsEmpty_ ) {
	createInteriorCells_();
    }
    return interior_cells_;
}


void DeviceGrid::createBoundaryFaces_() const {
    // we use the face_cells_ array to check if both face_cells are cells
    // If face f is a boundary face, then 
    // face_cells_[2 * f] or face_cells_[2 * f + 1] contains -1.
    
    // Launch a kernel where we use number_of_faces number of threads.
    // Use a 1D kernel for simplicity.
    // Assume that we do not need more blocks than available.
    kernelSetup s(number_of_faces_);

    // Create a vector of size number_of_faces_:
    thrust::device_vector<int> b_faces(number_of_faces_);
    // Fill it with the value number_of_faces_
    //     this is an illigal faca index
    thrust::fill(b_faces.begin(), b_faces.end(), number_of_faces_);
    int* b_faces_ptr = thrust::raw_pointer_cast( &b_faces[0] );
    boundaryFacesKernel<<<s.grid, s.block>>>( b_faces_ptr,
					      face_cells_,
					      number_of_faces_);
    
    // Remove unchanged values
    // See  - thrust::remove_if documentation 
    //      - the saxpy example in the algorithm chapter of the thrust pdf
    thrust::device_vector<int>::iterator new_end = thrust::remove_if(thrust::device, 
								     b_faces.begin(),
								     b_faces.end(),
								     unchanged(number_of_faces_));
    
    // new_end points now to where the legal values end,
    // but the vector still has size equal to number_of_faces_
    boundary_faces_ = CollOfFace(b_faces.begin(), new_end);
    boundaryFacesEmpty_ = false;
}


void DeviceGrid::createInteriorFaces_() const {
    // we use the face_cells_ array to check if both face_cells are cells
    // If face f is an interior face, then neither of
    // face_cells_[2 * f] nor face_cells_[2 * f + 1] contains -1.
    
    // Launch a kernel where we use number_of_faces number of threads.
    // Use a 1D kernel for simplicity.
    // Assume that we do not need more blocks than available.
     kernelSetup s(number_of_faces_);

    // Create a vector of size number_of_faces_:
    thrust::device_vector<int> i_faces(number_of_faces_);
    // Fill it with the value number_of_faces_
    //     this is an illigal faca index
    thrust::fill(i_faces.begin(), i_faces.end(), number_of_faces_);
    int* i_faces_ptr = thrust::raw_pointer_cast( &i_faces[0] );
    interiorFacesKernel<<<s.grid, s.block>>>( i_faces_ptr,
					      face_cells_,
					      number_of_faces_);
    // Remove unchanged values
    // See  - thrust::remove_if documentation 
    //      - the saxpy example in the algorithm chapter of the thrust pdf
    thrust::device_vector<int>::iterator new_end = thrust::remove_if(thrust::device, 
								     i_faces.begin(),
								     i_faces.end(),
								     unchanged(number_of_faces_));
    
    // new_end points now to where the legal values end,
    // but the vector still has size equal to number_of_faces_    
    interior_faces_ = CollOfFace(i_faces.begin(), new_end);
    interiorFacesEmpty_ = false;
}


// BOUNDARY CELLS
void DeviceGrid::createBoundaryCells_() const {
    // Returns a Collection of indices of boundary cells.
    // Algorithm:
    // for each cell c
    //     for (face f_index in [cell_facepos[c] : cell_facepos[c+1] )
    //          f = cell_faces[f_index]
    //          if ( face_cells[2*f] == -1 or face_cells[2*f + 1] == -1 )
    //              c is a boundary cell.

    // Kernel of number_of_cells_ threads
    // Operate on vector filled with number_of_cells_
    // Set cell index if boundary cell
    // Remove all elements equal to number_of_cells_.

    kernelSetup s(number_of_cells_);
    thrust::device_vector<int> b_cells(number_of_cells_);
    thrust::fill(b_cells.begin(), b_cells.end(), number_of_cells_);
    int* b_cells_ptr = thrust::raw_pointer_cast( &b_cells[0] );
    boundaryCellsKernel<<<s.grid, s.block>>>( b_cells_ptr,
					      number_of_cells_,
					      cell_facepos_,
					      cell_faces_,
					      face_cells_);

    // Remove values which still are number_of_cells_
    thrust::device_vector<int>::iterator new_end = thrust::remove_if(thrust::device,
								     b_cells.begin(),
								     b_cells.end(),
								     unchanged(number_of_cells_));
    boundary_cells_ = CollOfCell(b_cells.begin(), new_end);
    boundaryCellsEmpty_ = false;
}


// INTERIOR CELLS
void DeviceGrid::createInteriorCells_() const {
    // Same as boundaryCells, but the kernel is the other way around
    kernelSetup s(number_of_cells_);
    thrust::device_vector<int> i_cells(number_of_cells_);
    thrust::fill(i_cells.begin(), i_cells.end(), number_of_cells_);
    int* i_cells_ptr = thrust::raw_pointer_cast( &i_cells[0] );
    interiorCellsKernel<<<s.grid, s.block>>>( i_cells_ptr,
					      number_of_cells_,
					      cell_facepos_,
					      cell_faces_,
					      face_cells_);

    // Remove values which still are number_of_cells_
    thrust::device_vector<int>::iterator new_end = thrust::remove_if(thrust::device,
								     i_cells.begin(),
								     i_cells.end(),
								     unchanged(number_of_cells_));
    interior_cells_ = CollOfCell(i_cells.begin(), new_end);
    interiorCellsEmpty_ = false;
}


// FIRST AND SECOND
CollOfCell DeviceGrid::firstCell(CollOfFace coll) const {
    // The out collection will be of same size as the in collection

    // FirstCells are found from the face_cells_ array
    // for face f
    //     first(f) = face_cells_[2*f]
    
    // setup how many threads/blocks we need:
    kernelSetup s(coll.size());

    // create a vector of size number_of_faces_:
    thrust::device_vector<int> first(coll.size());
    int* first_ptr = thrust::raw_pointer_cast( &first[0] );
    if (coll.isFull()) {
	firstCellKernel<<<s.grid, s.block>>>( first_ptr, coll.size(), face_cells_);
    } else {
	int* index_ptr = coll.raw_pointer();
 	firstCellSubsetKernel<<<s.grid, s.block>>>( first_ptr, coll.size(),
						    index_ptr, face_cells_);
    }					
    return CollOfCell(first);
}

CollOfCell DeviceGrid::secondCell(CollOfFace coll) const {
    // SecondCells are found from the face_cells_ array
    // for face f
    //     second(f) = face_cells_[2*f + 1]

    // setup how many threads/blocks we need:
    kernelSetup s(coll.size());
    
    // create a vector of size number_of_faces_:
    thrust::device_vector<int> second(coll.size());
    int* second_ptr = thrust::raw_pointer_cast( &second[0] );
    if ( coll.isFull() ) {
	secondCellKernel<<<s.grid, s.block>>>( second_ptr, coll.size(), face_cells_);
    } else {
	secondCellSubsetKernel<<<s.grid, s.block>>>( second_ptr, coll.size(),
						     coll.raw_pointer(), face_cells_);
    }
    return CollOfCell(second);
}


// ----- NORM ----

CollOfScalar DeviceGrid::norm_of_cells(const thrust::device_vector<int>& cells,
				       const bool full) const {
    if (full) {
	CollOfScalar out(number_of_cells_);
	cudaStatus_ = hipMemcpy( out.data(), cell_volumes_, 
				  sizeof(double)*number_of_cells_,
				  hipMemcpyDeviceToDevice);
	checkError_("hipMemcpy in DeviceGrid::norm_of_cells");
	return CollOfScalar(std::move(out));
    }
    else {
	CollOfScalar out(cells.size());
	kernelSetup s = out.setup();
	const int* cells_ptr = thrust::raw_pointer_cast( &cells[0] );
	normKernel<<<s.grid, s.block>>>( out.data(), cells_ptr, cells.size(),
					 cell_volumes_);
	return CollOfScalar(std::move(out));
    }
}

CollOfScalar DeviceGrid::norm_of_faces(const thrust::device_vector<int>& faces,
				       const bool full) const {
    if (full) {
	CollOfScalar out(number_of_faces_);
	cudaStatus_ = hipMemcpy(out.data(), face_areas_, 
				 sizeof(double)*number_of_faces_,
				 hipMemcpyDeviceToDevice);
	checkError_("hipMemcpy in DeviceGrid::norm_of_cells");
	return CollOfScalar(std::move(out));
    }
    else {
	CollOfScalar out(faces.size());
	kernelSetup s = out.setup();
	const int* faces_ptr = thrust::raw_pointer_cast( &faces[0] );
	normKernel<<<s.grid, s.block>>>( out.data(), faces_ptr, faces.size(),
					 face_areas_);
	return CollOfScalar(std::move(out));
    }
}


// CENTROID

CollOfVector DeviceGrid::centroid(const thrust::device_vector<int>& indices,
				  const bool full,
				  const int codim) const {
    if (full) {
	if (codim == 0) { // All cells
	    CollOfVector out(number_of_cells_, dimensions_);
	    cudaStatus_ = hipMemcpy(out.data(), cell_centroids_,
				     sizeof(double)*dimensions_*number_of_cells_,
				     hipMemcpyDeviceToDevice);
	    checkError_("hipMemcpy in DeviceGrid::centroid(..) -> full -> codim=0");
	    return CollOfVector(std::move(out));
	}
	else { // All faces
	    CollOfVector out(number_of_faces_, dimensions_);
	    cudaStatus_ = hipMemcpy(out.data(), face_centroids_,
				     sizeof(double)*dimensions_*number_of_faces_,
				     hipMemcpyDeviceToDevice);
	    checkError_("hipMemcpy in DeviceGrid::centroids(..) -> full -> codim=1");
	    return CollOfVector(std::move(out));
	}
    }
    else {
	CollOfVector out(indices.size(), dimensions_);
	// Set up a kernel to find the subset
	// Easy implementation: 
	// CollOfVector::block() and grid() assumes one thread per double value
	// Our kernel use one thread per vector, so we overshoot a bit.
	kernelSetup s = out.element_setup();
	const int* indices_ptr = thrust::raw_pointer_cast( &indices[0] );
	
	// Get a pointer to the correct set of centroids:
	const double* all_centroids = cell_centroids_;
	if ( codim == 1) {
	    all_centroids = face_centroids_;
	}
	centroidKernel<<<s.grid, s.block>>>( out.data(),
					     indices_ptr,
					     all_centroids,
					     out.numVectors(),
					     dimensions_);
	return CollOfVector(std::move(out));
    }
}


// NORMAL:
CollOfVector DeviceGrid::normal( const CollOfFace& faces) const {
    CollOfVector out(faces.size(), dimensions_);
    // hipMemcpy to get the normals if the set is full.
    if ( faces.isFull() ) {
	
	cudaStatus_ = hipMemcpy(out.data(), face_normals_,
				 sizeof(double)*out.numVectors()*dimensions_,
				 hipMemcpyDeviceToDevice);
	checkError_("hipMemcpy(face_normals) in DeviceGrid::normal(const CollOfFaces&)");
    }
    else {
	// Need a Kernel to fetch only the correct ones.
	// Easy implementation:
	// CollOfVector::block() and grid() assumes one thread per double value
	// Our kernel use one thread per vector, so we overshoot a bit.
	kernelSetup s = out.element_setup();
	faceNormalsKernel<<<s.grid, s.block>>>(out.data(),
					       faces.raw_pointer(),
					       face_normals_,
					       out.numVectors(),
					       dimensions_);
    }
    return CollOfVector(std::move(out));
}



// ----------- GET FUNCTIONS! ------------------

int DeviceGrid::dimensions() const {
    return dimensions_;
}

int DeviceGrid::number_of_cells() const {
    return number_of_cells_;
}

int DeviceGrid::number_of_faces() const {
    return number_of_faces_;
}

int* DeviceGrid::cell_facepos() const {
    return cell_facepos_;
}

int* DeviceGrid::cell_faces() const {
    return cell_faces_;
}

int* DeviceGrid::face_cells() const {
    return face_cells_;
}

// ---------- ERROR CHECKING! -----------------------

// Check if for CUDA error and throw OPM exception if there is one.
void DeviceGrid::checkError_(const std::string& msg) const {
    if ( cudaStatus_ != hipSuccess ) {
	OPM_THROW(std::runtime_error, "\nCuda error\n\t" << msg << " - Error code: " << hipGetErrorString(cudaStatus_));
    }        
}




// ----------- GRID KERNELS -------------------------

__global__ void wrapDeviceGrid::boundaryFacesKernel( int* b_faces,
						     const int* face_cells,
						     const int number_of_faces) 
{
    const int face = myID();
    if (face < number_of_faces) {
	if ( (face_cells[2*face] == -1) || (face_cells[2*face + 1] == -1) ) {
	    b_faces[face] = face;
	}
    }
}


__global__ void wrapDeviceGrid::interiorFacesKernel( int* i_faces,
						     const int* face_cells,
						     const int number_of_faces)
{
    const int face = myID();
    if ( face < number_of_faces) {
	if ( (face_cells[2*face] != -1) && (face_cells[2*face + 1] != -1) ) {
	    i_faces[face] = face;
	}
    }
}


__global__ void wrapDeviceGrid::boundaryCellsKernel(int* b_cells,
						    const int number_of_cells,
						    const int* cell_facepos,
						    const int* cell_faces,
						    const int* face_cells)
{
    const int cell = myID();
    if ( cell < number_of_cells) {
	bool boundary = false;
	int face;
	for ( int f_i = cell_facepos[cell]; f_i < cell_facepos[cell + 1]; f_i++) {
	    face = cell_faces[f_i];
	    if ( (face_cells[ 2*face ] == -1) || (face_cells[ 2*face +1] == -1) ) {
		boundary = true;
	    }
	}
	if (boundary) {
	    b_cells[cell] = cell;
	}
    }
}


__global__ void wrapDeviceGrid::interiorCellsKernel( int* i_cells,
						     const int number_of_cells,
						     const int* cell_facepos,
						     const int* cell_faces,
						     const int* face_cells)
{
    const int cell = myID();
    if ( cell < number_of_cells) {
	bool interior = true;
	int face;
	for ( int f_i = cell_facepos[cell]; f_i < cell_facepos[cell + 1]; f_i++) {
	    face = cell_faces[f_i];
	    if ( (face_cells[ 2*face ] == -1) || (face_cells[ 2*face +1] == -1) ) {
		interior = false;
	    }
	}
	if ( interior ) {
	    i_cells[cell] = cell;
	}
    }

}


__global__ void wrapDeviceGrid::firstCellKernel( int* first,
						 const int number_of_faces,
						 const int* face_cells)
{
    // For face f:
    //     first(f) = face_cells[2*f]
    const int face = myID();
    if ( face < number_of_faces ) {
	first[face] = face_cells[2*face];
    }
}

__global__ void wrapDeviceGrid::firstCellSubsetKernel( int* first,
						       const int number_of_faces,
						       const int* face_index,
						       const int* face_cells)
{
    // For thread i:
    //      first(i) = face_cells[2*face_index[i]]
    const int index = myID();
    if ( index < number_of_faces ) {
	first[index] = face_cells[2*face_index[index]];
    }
}

__global__ void wrapDeviceGrid::secondCellKernel( int* second,
						  const int number_of_faces,
						  const int* face_cells)
{
    // For face f:
    //     second(f) = face_cells[2*f + 1]
    const int face = myID();
    if ( face < number_of_faces ) {
	second[face] = face_cells[2*face + 1];
    }
 }

__global__ void wrapDeviceGrid::secondCellSubsetKernel( int* second,
							const int number_of_faces,
							const int* face_index,
							const int* face_cells)
{
    // for thread i
    //     second[i] = face_cells[2*face_index[i] + 1]
    const int index = myID();
    if ( index < number_of_faces ) {
	second[index] = face_cells[2*face_index[index] + 1];
    }
}


// NORM KERNEL


__global__ void wrapDeviceGrid::normKernel( double* out,
					    const int* indices,
					    const int out_size,
					    const double* norm_values) 
{
    const int index = myID();
    if ( index < out_size ) {
	out[index] = norm_values[indices[index]];
    }
}

// CENTROID KERNEL

__global__ void wrapDeviceGrid::centroidKernel( double* out,
						const int* subset_indices,
						const double* all_centroids,
						const int num_vectors,
						const int dimensions)
{
    // EASY IMPLEMENTATION:
    // One thread for each vector
    const int vec_id = myID();
    if ( vec_id < num_vectors ) {
	const int cell_index = subset_indices[vec_id];
	// Iterating over the element in the vector we create
	for (int i = 0; i < dimensions; i++) {
	    out[vec_id*dimensions + i] = all_centroids[cell_index * dimensions + i];
	}
    }
}



// FACE NORMALS
__global__ void wrapDeviceGrid::faceNormalsKernel( double* out,
						   const int* faces,
						   const double* all_face_normals,
						   const int num_vectors,
						   const int dimensions)
{
    // EASY IMPLEMENTATION
    // One thread for each vector
    const int vec_id = myID();
    if ( vec_id < num_vectors ) {
	const int face_id = faces[vec_id];
	for( int i = 0; i < dimensions; i++) {
	    out[vec_id*dimensions + i] = all_face_normals[face_id*dimensions + i];
	}
    }
}



// NORMALIZE FACE NORMALS
__global__ void wrapDeviceGrid::normalizeAllFaceNormals( double* normals, 
                                                         const double* face_areas_,
                                                         const int num_vectors,
                                                         const int dimensions)
{
    // One thread for each vector
    const int face_id = myID();
    if ( face_id < num_vectors ) {
        if(dimensions == 1){
            normals[face_id] /= face_areas_[face_id];
        } else
        if(dimensions == 2){
            normals[face_id] /= face_areas_[face_id];
            normals[face_id*dimensions+1] /= face_areas_[face_id];
        } else
        if(dimensions == 3) {
            normals[face_id*dimensions]   /= face_areas_[face_id];
            normals[face_id*dimensions+1] /= face_areas_[face_id];
            normals[face_id*dimensions+2] /= face_areas_[face_id];
        }
    }
}
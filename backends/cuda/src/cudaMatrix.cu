#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

#include <thrust/device_vector.h>
#include <thrust/detail/raw_pointer_cast.h>

#include <opm/common/ErrorMacros.hpp>

#include <vector>
#include <iostream>
#include <string>

#include <Eigen/Sparse>

#include "CudaMatrix.hpp"
#include "CudaArray.hpp" // kernels for scalar multiplications
#include "CollOfScalar.hpp" // for constructor for diagonal matrix.
#include "equelleTypedefs.hpp"
#include "device_functions.cuh"
#include "CusparseManager.hpp"

using namespace equelleCUDA;
using namespace wrapCudaMatrix;

using std::vector;

// Implementation of member functions of CudaMatrix

// Default constructor:
CudaMatrix::CudaMatrix() 
    : rows_(0),
      cols_(0),
      nnz_(0),
      csrVal_(0),
      csrRowPtr_(0),
      csrColInd_(0),
      sparseStatus_(HIPSPARSE_STATUS_SUCCESS),
      cudaStatus_(hipSuccess),
      description_(0),
      operation_(HIPSPARSE_OPERATION_NON_TRANSPOSE),
      diagonal_(false)
{
    createGeneralDescription_("CudaMatrix::CudaMatrix()");
}


// Constructor from host values:
CudaMatrix::CudaMatrix( const double* val, const int* rowPtr, const int* colInd,
      const int nnz, const int rows, const int cols)
    : rows_(rows),
      cols_(cols),
      nnz_(nnz),
      csrVal_(0),
      csrRowPtr_(0),
      csrColInd_(0),
      sparseStatus_(HIPSPARSE_STATUS_SUCCESS),
      cudaStatus_(hipSuccess),
      description_(0),
      operation_(HIPSPARSE_OPERATION_NON_TRANSPOSE),
      diagonal_(false)
{
    // Allocate memory
    allocateMemory("CudaMatrix host constructor");

    // Copy data:
    cudaStatus_ = hipMemcpy( csrVal_, val, nnz_*sizeof(double), 
            hipMemcpyHostToDevice);
    checkError_("hipMemcpy(csrVal_) in CudaMatrix host constructor");

    cudaStatus_ = hipMemcpy( csrRowPtr_, rowPtr, (rows_ + 1)*sizeof(int),
            hipMemcpyHostToDevice);
    checkError_("hipMemcpy(csrRowPtr_) in CudaMatrix host constructor");

    cudaStatus_ = hipMemcpy( csrColInd_, colInd, nnz_*sizeof(int),
            hipMemcpyHostToDevice);
    checkError_("hipMemcpy(csrColInd_) in CudaMatrix host constructor");

    createGeneralDescription_("CudaMatrix host constructor");
}


// Constructor from Eigen Matrix
CudaMatrix::CudaMatrix(const Eigen::SparseMatrix<Scalar>& eigen)
    : rows_(eigen.rows()),
      cols_(eigen.cols()),
      nnz_(eigen.nonZeros()),
      csrVal_(0),
      csrRowPtr_(0),
      csrColInd_(0),
      sparseStatus_(HIPSPARSE_STATUS_SUCCESS),
      cudaStatus_(hipSuccess),
      description_(0),
      operation_(HIPSPARSE_OPERATION_NON_TRANSPOSE),
      diagonal_(false)
{
    // Should have a check here to ensure that the matrix is stored 
    // in a row-major format.
    
    // Opm::HelperOps creates helper matrices in column major format.
    // Copy the input to a row major matrix instead:
    Eigen::SparseMatrix<Scalar, Eigen::RowMajor> rowmajor(eigen);

    // Allocate memory:
    allocateMemory("CudaMatrix Eigen constructor");

    // Copy arrays:
    cudaStatus_ = hipMemcpy( csrVal_, rowmajor.valuePtr(), nnz_*sizeof(double),
            hipMemcpyHostToDevice);
    checkError_("hipMemcpy(csrVal_) in CudaMatrix Eigen constructor");
    cudaStatus_ = hipMemcpy( csrRowPtr_, rowmajor.outerIndexPtr(), (rows_+1)*sizeof(int),
            hipMemcpyHostToDevice);
    checkError_("hipMemcpy(csrRowPtr_) in CudaMatrix Eigen constructor");
    cudaStatus_ = hipMemcpy( csrColInd_, rowmajor.innerIndexPtr(), nnz_*sizeof(int),
            hipMemcpyHostToDevice);
    checkError_("hipMemcpy(csrColInd_) in CudaMatrix Eigen constructor");

    createGeneralDescription_("CudaMatrix Eigen constructor");
} // constructor from Eigen


// Identity matrix constructor
CudaMatrix::CudaMatrix(const int size) 
    : rows_(size),
      cols_(size),
      nnz_(size),
      csrVal_(0),
      csrRowPtr_(0),
      csrColInd_(0),
      sparseStatus_(HIPSPARSE_STATUS_SUCCESS),
      cudaStatus_(hipSuccess),
      description_(0),
      operation_(HIPSPARSE_OPERATION_NON_TRANSPOSE),
      diagonal_(true)
{
    // Allocate memory:
    allocateMemory("CudaMatrix identity matrix constructor");

    // Call a kernel that writes the correct data:
    kernelSetup s(size+1);
    initIdentityMatrix<<<s.grid, s.block>>>(csrVal_, csrRowPtr_, csrColInd_, nnz_);

    createGeneralDescription_("CudaMatrix identity matrix constructor");
}

// Constructs a matrix of size rows*cols with nnz non-zero elements.
// The constructor allocates memory, but does not initialize it.
CudaMatrix::CudaMatrix(const int rows, const int cols, const int nnz)
    : rows_(rows),
      cols_(cols),
      nnz_(nnz),
      csrVal_(0),
      csrRowPtr_(0),
      csrColInd_(0),
      sparseStatus_(HIPSPARSE_STATUS_SUCCESS),
      cudaStatus_(hipSuccess),
      description_(0),
      operation_(HIPSPARSE_OPERATION_NON_TRANSPOSE),
      diagonal_(false)
{
    allocateMemory("CudaMatrix(rows, cols, nnz)");

    createGeneralDescription_("CudaMatrix(rows, cols, nnz)");
}

// Constructor for creating a diagonal matrix from the value of a CollOfScalar
CudaMatrix::CudaMatrix(const CollOfScalar& coll)
    : rows_(coll.size()),
      cols_(rows_),
      nnz_(rows_),
      csrVal_(0),
      csrRowPtr_(0),
      csrColInd_(0),
      sparseStatus_(HIPSPARSE_STATUS_SUCCESS),
      cudaStatus_(hipSuccess),
      description_(0),
      operation_(HIPSPARSE_OPERATION_NON_TRANSPOSE),
      diagonal_(true)
{
    // Allocate memory:
    allocateMemory("CudaMatrix diagonal matrix constructor");

    // Call a kernel to write the correct data:
    kernelSetup s(nnz_+1);
    initDiagonalMatrix<<<s.grid, s.block>>>(csrVal_, csrRowPtr_, csrColInd_, coll.data(),
              nnz_);
    
    createGeneralDescription_("CudaMatrix diagonal matrix constructor");
}

// Move constructor
CudaMatrix::CudaMatrix(CudaMatrix&& mat)
    : rows_(mat.rows_),
      cols_(mat.cols_),
      nnz_(mat.nnz_),
      csrVal_(0),
      csrRowPtr_(0),
      csrColInd_(0),
      sparseStatus_(HIPSPARSE_STATUS_SUCCESS),
      cudaStatus_(hipSuccess),
      description_(0),
      operation_(mat.operation_),
      diagonal_(mat.diagonal_)
{
    swap(mat);
    createGeneralDescription_("CudaMatrix move constructor");
}

// Constructor for creating a diagonal matrcit from a CudaArray
CudaMatrix::CudaMatrix(const CudaArray& array)
    : rows_(array.size()),
      cols_(rows_),
      nnz_(rows_),
      csrVal_(0),
      csrRowPtr_(0),
      csrColInd_(0),
      sparseStatus_(HIPSPARSE_STATUS_SUCCESS),
      cudaStatus_(hipSuccess),
      description_(0),
      operation_(HIPSPARSE_OPERATION_NON_TRANSPOSE),
      diagonal_(true)
{
    // Allocate memory:
    allocateMemory("CudaMatrix::CudaMatrix(CudaArray)");

    // Call a kerenl to write the correct data:
    kernelSetup s(nnz_ + 1);
    initDiagonalMatrix<<<s.grid, s.block>>>(csrVal_, csrRowPtr_, csrColInd_,
              array.data(), nnz_);
    
    createGeneralDescription_("CudaMatrix::CudaMatrix(CudaArray)");
}
              

// Constructor for diagonal from booleans
CudaMatrix::CudaMatrix(const CollOfBool& bools) 
    : rows_(bools.size()),
      cols_(rows_),
      nnz_(rows_),
      csrVal_(0),
      csrRowPtr_(0),
      csrColInd_(0),
      sparseStatus_(HIPSPARSE_STATUS_SUCCESS),
      cudaStatus_(hipSuccess),
      description_(0),
      operation_(HIPSPARSE_OPERATION_NON_TRANSPOSE),
      diagonal_(true)
{ 
    allocateMemory("CudaMatrix::CudaMatrix(CollOfBool)");
    
    kernelSetup s(nnz_ + 1);
    const bool* bool_ptr = thrust::raw_pointer_cast( &bools[0] );
    initBooleanDiagonal<<<s.grid, s.block>>>( csrVal_, csrRowPtr_, csrColInd_,
                bool_ptr, rows_);
    
    createGeneralDescription_("CudaMatrix::CudaMatrix(CollOfBool)");
}

// Restriction matrix constructor:
CudaMatrix::CudaMatrix(const thrust::device_vector<int> set,
           const int full_size) 
    : rows_(set.size()),
      cols_(full_size),
      nnz_(rows_),
      csrVal_(0),
      csrRowPtr_(0),
      csrColInd_(0),
      sparseStatus_(HIPSPARSE_STATUS_SUCCESS),
      cudaStatus_(hipSuccess),
      description_(0),
      operation_(HIPSPARSE_OPERATION_NON_TRANSPOSE),
      diagonal_(false)
{
    // Allocate memory:
    allocateMemory("CudaMatrix constructor for On from full set");
    
    // Matrix is flat, more cols than rows.
    //   - each row has one element, hence csrRowPtr = [0,1,2,...,rows_] (size rows+1)
    //   - all nnz elements are 1, hence csrVal = [1,1,1,...,1] (size rows)
    //   - csrColInd = to_set (size rows)
    const int* set_ptr = thrust::raw_pointer_cast( &set[0] );
    kernelSetup s(rows_ + 1);
    initRestrictionMatrix<<<s.grid, s.block>>>( csrVal_, csrRowPtr_, csrColInd_,
            set_ptr, rows_);

    createGeneralDescription_("CudaMatrix constructor for On from full set");
}



// Copy constructor:
CudaMatrix::CudaMatrix(const CudaMatrix& mat)
    : rows_(mat.rows_),
      cols_(mat.cols_),
      nnz_(mat.nnz_),
      csrVal_(0),
      csrRowPtr_(0),
      csrColInd_(0),
      sparseStatus_(HIPSPARSE_STATUS_SUCCESS),
      cudaStatus_(hipSuccess),
      description_(0),
      operation_(mat.operation_),
      diagonal_(mat.diagonal_)
{
    // Copy arrays if they exist:
    if ( mat.csrVal_ != 0 ) {
  cudaStatus_ = hipMalloc( (void**)&csrVal_, nnz_*sizeof(double));
  checkError_("hipMalloc(csrVal_) in CudaMatrix copy constructor");
  cudaStatus_ = hipMemcpy( csrVal_, mat.csrVal_, nnz_*sizeof(double),
          hipMemcpyDeviceToDevice);
  checkError_("hipMemcpy(csrVal_) in CudaMatrix copy constructor");
    }
    if ( mat.csrRowPtr_ != 0 ) {
  cudaStatus_ = hipMalloc( (void**)&csrRowPtr_, (rows_+1)*sizeof(int));
  checkError_("hipMalloc(csrRowPtr_) in CudaMatrix copy constructor");
  cudaStatus_ = hipMemcpy( csrRowPtr_, mat.csrRowPtr_, (rows_+1)*sizeof(int),
          hipMemcpyDeviceToDevice);
  checkError_("hipMemcpy(csrRowPtr_) in CudaMatrix copy constructor");
    }
    if ( mat.csrColInd_ != 0 ) {
  cudaStatus_ = hipMalloc( (void**)&csrColInd_, nnz_*sizeof(int));
  checkError_("hipMalloc(csrColInd_) in CudaMalloc copy constructor");
  cudaStatus_ = hipMemcpy( csrColInd_, mat.csrColInd_, nnz_*sizeof(int),
          hipMemcpyDeviceToDevice);
  checkError_("hipMemcpy(csrColInd_) in CudaMatrix copy constructor");
    }
    
    createGeneralDescription_("CudaMatrix copy constructor");
}


// Copy assignment operator:
CudaMatrix& CudaMatrix::operator= (const CudaMatrix& other) {

    // Protect against self assignment
    if ( this != &other ) {
  
  if ( !other.isEmpty() ) {
      
      // Check if we have to reallocate memory depending on nnz:
      if ( nnz_ != other.nnz_ ) { 
    nnz_ = other.nnz_;
    
    // Free and reallocate csrVal_, but only free if csrVal_ != 0
    if ( csrVal_ != 0 ) {
        cudaStatus_ = hipFree(csrVal_);
        checkError_("hipFree(csrVal_) in CudaMatrix copy assignment operator");
    }
    cudaStatus_ = hipMalloc( (void**)&csrVal_, nnz_*sizeof(double));
    checkError_("hipMalloc(csrVal_) in CudaMatrix copy assignment operator");
    
    // Free (if nonzero) and allocate csrColInd_
    if ( csrColInd_ != 0 ) {
        cudaStatus_ = hipFree(csrColInd_);
        checkError_("hipFree(csrColInd_) in CudaMatrix copy assignment operator");
    }
    cudaStatus_ = hipMalloc( (void**)&csrColInd_, nnz_*sizeof(int));
    checkError_("hipMalloc(csrColInd_) in CudaMatrix copy assignment operator");
      } // if (nnz != other.nnz_)
      
      // Check if we have to reallocate memory depending on rows:
      if ( rows_ != other.rows_ ) {
    rows_ = other.rows_;
    if ( csrRowPtr_ != 0 ) {
        cudaStatus_ = hipFree(csrRowPtr_);
        checkError_("hipFree(csrRowPtr_) in CudaMatrix copy assignment operator");
    }
    cudaStatus_ = hipMalloc( (void**)&csrRowPtr_, (rows_+1)*sizeof(int));
    checkError_("hipMalloc(csrRowPtr_) in CudaMatrix copy assignment operator");
      } // if ( rows_ != other.rows_ )
      
      cols_ = other.cols_;
      
      // All arrays correct sizes. Copy data:
      cudaStatus_ = hipMemcpy( csrVal_, other.csrVal_, nnz_*sizeof(double),
              hipMemcpyDeviceToDevice);
      checkError_("hipMemcpy(csrVal_) in CudaMatrix copy assignment operator");
      cudaStatus_ = hipMemcpy( csrRowPtr_, other.csrRowPtr_, (rows_+1)*sizeof(int),
              hipMemcpyDeviceToDevice);
      checkError_("hipMemcpy(csrRowPtr_) in CudaMatrix copy assignment operator");
      cudaStatus_ = hipMemcpy( csrColInd_, other.csrColInd_, nnz_*sizeof(int),
              hipMemcpyDeviceToDevice);
      checkError_("hipMemcpy(csrColInd_) in CudaMatrix copy assignment operator");
      
  } // other is not empty
  else {
      // Other is empty!
      if ( !this->isEmpty() ) {
    if ( csrVal_ != 0 ) {
        cudaStatus_ = hipFree(csrVal_);
        checkError_("hipFree(csrVal_) in CudaMatrix copy assignment for empty other");
        csrVal_ = 0;
    }
    if ( csrRowPtr_ != 0 ) {
        cudaStatus_ = hipFree(csrRowPtr_);
        checkError_("hipFree(csrRowPtr_) in CudaMatrix copy assignment for empty other");
        csrRowPtr_ = 0;
    }
    if ( csrColInd_ != 0 ) {
        cudaStatus_ = hipFree(csrColInd_);
        checkError_("hipFree(csrColInd_) in CudaMatrix copy assignment for empty other");
        csrColInd_ = 0;
    }
    nnz_ = 0;
    rows_ = 0;
    cols_ = 0;
      }
       

  } // if other is empty
  
  // Do not have to care about description, as it is the same for all matrices!
  operation_ = other.operation_;
  diagonal_ = other.diagonal_;

    } // if ( this != &other)
    
    return *this;
}


// Move assignment operator
CudaMatrix& CudaMatrix::operator=(CudaMatrix&& other)
{
    swap(other);
    return *this;
}


// Swap function used for move semantics
void CudaMatrix::swap(CudaMatrix& other) noexcept
{

    std::swap(nnz_, other.nnz_);
    std::swap(csrVal_, other.csrVal_);
    std::swap(csrColInd_, other.csrColInd_);
    std::swap(rows_, other.rows_);
    std::swap(csrRowPtr_, other.csrRowPtr_);
    std::swap(cols_, other.cols_);
    
    operation_ = other.operation_;
    diagonal_ = other.diagonal_;
}


// Destructor
CudaMatrix::~CudaMatrix() {
    // Free pointers if not zero.
    if ( csrVal_ != 0 ) {
  cudaStatus_ = hipFree(csrVal_);
  checkError_("hipFree(csrVal_) in CudaMatrix::~CudaMatrix");
    }
    if ( csrRowPtr_ != 0 ) {
  cudaStatus_ = hipFree(csrRowPtr_);
  checkError_("hipFree(csrRowPtr_) in CudaMatrix::~CudaMatrix");
    }
    if ( csrColInd_ != 0 ) {
  cudaStatus_ = hipFree(csrColInd_);
  checkError_("hipFree(csrColInd_) in CudaMatrix::~CudaMatrix");
    }

    // Destroy description_ 
    sparseStatus_ = hipsparseDestroyMatDescr( description_ );
    checkError_("hipsparseDestroyMatDescr() in CudaMatrix::~CudaMatrix()");

}





int CudaMatrix::nnz() const {
    return nnz_;
}
int CudaMatrix::rows() const {
    return rows_;
}
int CudaMatrix::cols() const {
    return cols_;
}

bool CudaMatrix::isEmpty() const {
    return (csrVal_ == NULL);
}

const double* CudaMatrix::csrVal() const {
    return csrVal_;
}
const int* CudaMatrix::csrRowPtr() const {
    return csrRowPtr_;
}
const int* CudaMatrix::csrColInd() const {
    return csrColInd_;
}
double* CudaMatrix::csrVal() {
    return csrVal_;
}
int* CudaMatrix::csrRowPtr() {
    return csrRowPtr_;
}
int* CudaMatrix::csrColInd() {
    return csrColInd_;
}



// Copy to host:
hostMat CudaMatrix::toHost() const {
    if ( (csrVal_ == 0) || (csrRowPtr_ == 0) || (csrColInd_ == 0) ) {
  OPM_THROW(std::runtime_error, "Tried to copy matrix to host, but the pointers are (" << csrVal_ << "," << csrRowPtr_ << "," << csrColInd_ << ")");
    }

    vector<double> v(nnz_, -1);
    cudaStatus_ = hipMemcpy( &v[0], csrVal_, nnz_*sizeof(double),
            hipMemcpyDeviceToHost);
    checkError_("hipMemcpy(csrVal_) in CudaMatrix::toHost()");
    
    vector<int> rp(rows_+1, -1);
    cudaStatus_ = hipMemcpy( &rp[0], csrRowPtr_, (rows_+1)*sizeof(int),
            hipMemcpyDeviceToHost);
    checkError_("hipMemcpy(csrRowPtr_) in CudaMatrix::toHost()");
    
    vector<int> ci(nnz_, -1);
    cudaStatus_ = hipMemcpy( &ci[0], csrColInd_, nnz_*sizeof(int),
            hipMemcpyDeviceToHost);
    checkError_("hipMemcpy(csrColInd_) in CudaMatrix::toHost()");
    
    hostMat out;
    out.vals = v;
    out.rowPtr = rp;
    out.colInd = ci;
    out.nnz = nnz_;
    out.rows = rows_;
    out.cols = cols_;
    return out;
}



// TRANSPOSE
CudaMatrix CudaMatrix::transpose() const {
    CudaMatrix out = *this;
    out.operation_ = HIPSPARSE_OPERATION_TRANSPOSE;
    return CudaMatrix(std::move(out));
}


// Error checking:
void CudaMatrix::checkError_(const std::string& msg) const {
    if ( cudaStatus_ != hipSuccess) {
  OPM_THROW(std::runtime_error, "\nCuda error\n\t" << msg << " - Error code: "<< hipGetErrorString(cudaStatus_) );
    }
    if ( sparseStatus_ != HIPSPARSE_STATUS_SUCCESS ) {
  OPM_THROW(std::runtime_error, "\ncusparse error\n\t" << msg << " - Error code: " << sparseStatus_);
    }
}

void CudaMatrix::checkError_(const std::string& msg, const std::string& caller) const {
    checkError_(msg + caller);
}

void CudaMatrix::createGeneralDescription_(const std::string& msg) {
    sparseStatus_ = hipsparseCreateMatDescr( &description_ );
    checkError_("hipsparseCreateMatDescr() in " + msg);
    
    sparseStatus_ = hipsparseSetMatType(description_, HIPSPARSE_MATRIX_TYPE_GENERAL);
    checkError_("hipsparseSetMatType() in " + msg);
    sparseStatus_ = hipsparseSetMatIndexBase(description_, HIPSPARSE_INDEX_BASE_ZERO);
    checkError_("hipsparseSetMatIndexBase() in " + msg);
}


// MEMORY ALLOCATIONS
void CudaMatrix::allocateMemory(const std::string& caller) {
    // Error checking:
    if ( csrVal_ != 0 ) 
  OPM_THROW(std::runtime_error, "Error in CudaMatrix::allocateMemory\n" << "\tcsrVal_ already allocated.\n\tCalled from " << caller);
    if ( csrRowPtr_ != 0 ) {
  OPM_THROW(std::runtime_error, "Error in CudaMatrix::allocateMemory\n" << "\tcsrRowPtr_ already allocated.\n\tCalled from " << caller);
    }
    if ( csrColInd_ != 0 ) {
  OPM_THROW(std::runtime_error, "Error in CudaMatrix::allocateMemory\n" << "\tcsrColInd_ already allocated.\n\tCalled from " << caller);
    }
    
    // Allocating
    cudaStatus_ = hipMalloc( (void**)&csrVal_, nnz_*sizeof(double));
    checkError_("hipMalloc(csrVal_) in ", caller);
    cudaStatus_ = hipMalloc( (void**)&csrRowPtr_, (rows_+1)*sizeof(int));
    checkError_("hipMalloc(csrRowPtr_) in ", caller);
    cudaStatus_ = hipMalloc( (void**)&csrColInd_, nnz_*sizeof(int));
    checkError_("hipMalloc(csrColInd_) in ", caller);
}


// ERROR CHECKING FOR "CudaMatrix * CudaMatrix"
int CudaMatrix::confirmMultSize(const CudaMatrix& lhs, const CudaMatrix& rhs) {
    
    // We need to identify what are the true lhs sizes and rhs sizes wrt "transposity"

    int leftCols = lhs.cols_;
    int leftRows = lhs.rows_;
    int rightCols = rhs.cols_;
    int rightRows = rhs.rows_;
    
    if ( lhs.isTranspose() ) {
  leftCols = lhs.rows_;
  leftRows = lhs.cols_;
    }
    if ( rhs.isTranspose() ) {
  rightCols = rhs.rows_;
  rightRows = rhs.cols_;
    }

    // Check sizes
    if ( leftCols != rightRows ) {
  OPM_THROW(std::runtime_error, "Error in CudaMatrix * CudaMatrix size checking\n" << "\tMatrices of illegal sizes.\n" << "\tlhs.cols_ = " << leftCols << "\n\trhs.rows_ = " << rightRows);
    }

    // If test passed, assign this with correct rows and cols
    this->rows_ = leftRows;
    this->cols_ = rightCols;

    // Return inner size. 
    return leftCols;
}


bool CudaMatrix::isTranspose() const {
    return ( operation_ == HIPSPARSE_OPERATION_TRANSPOSE );
}


// --------------------- OVERLOADING OF OPERATORS -------------------------- //

// Operator +
CudaMatrix equelleCUDA::operator+(const CudaMatrix& lhs, const CudaMatrix& rhs) {
    // If one of the matrices is emtpy, we interpret it as a matrix filled with
    // zeros, and therefore just return the other matrix.
    // This is convenient when we implement autodiff by using CudaMatrix.
    if ( lhs.isEmpty() ) {
  return rhs;
    } 
    else if ( rhs.isEmpty() ) {
  return lhs;
    } 
    else {
  return CusparseManager::matrixAddition(lhs,rhs);
    }
}

CudaMatrix equelleCUDA::operator-(const CudaMatrix& lhs, const CudaMatrix& rhs) {
    // If one of the matrices is emtpy, we interpret it as a matrix filled with
    // zeros, and therefore just return the other matrix.
    // This is convenient when we implement autodiff by using CudaMatrix.
    if ( lhs.isEmpty() ) {
  return -1.0*rhs;
    }
    else if ( rhs.isEmpty() ) {
  return lhs;
    }
    else {
  return CusparseManager::matrixSubtraction(lhs, rhs);
    }
}


CudaMatrix equelleCUDA::operator*(const CudaMatrix& lhs, const CudaMatrix& rhs)
{
    // If any of them are empty, we return an empty matrix.
    // An empty matrix is interpreted as a correctly sized matrix of zeros.
    // This lets us not worry about empty derivatives for autodiff.
    if ( lhs.isEmpty() || rhs.isEmpty() ) {
        return CudaMatrix();
    }

    // Some functionality is implemented by multiplying with a diagonal matrix
    // from the left. Since csrGemm is a hotspot, we handle these cases more efficient
    // with this function:
    if ( lhs.diagonal_ ) {
        return lhs.diagonalMultiply(rhs);
    }

    return CusparseManager::matrixMultiply(lhs, rhs);
}


// Matrix * vector
CudaArray equelleCUDA::operator*(const CudaMatrix& mat, const CudaArray& vec) {
    //std::cout << "-------MATRIX * VECTOR ---------\n";

     // Check that sizes match - Depend on transpose matrix or not.
    int resultingVectorSize;
    if ( !mat.isTranspose() ) { // NOT transposed
  if ( mat.cols_ != vec.size() ) {
      OPM_THROW(std::runtime_error, "Error in matrix * vector operation as matrix is of size " << mat.rows_ << " by " << mat.cols_ << " and the vector of size " << vec.size());
  }
  resultingVectorSize = mat.rows_;
  //cols = mat.cols_;
    }
    else { // matrix IS transposed
  if ( mat.rows_ != vec.size() ) {
      OPM_THROW(std::runtime_error, "Error in transposed matrix * vector operation as matrix is of size " << mat.cols_ << " by " << mat.rows_ << " and the vector of size " << vec.size());
  }
  resultingVectorSize = mat.cols_;
    }

    
    // Call cusparse matrix-vector operation:
    // y = alpha*op(A)*x + beta*y
    // with alpha=1, beta=0, op=non_transpose
    CudaArray out(resultingVectorSize);
    const double alpha = 1.0;
    const double beta = 0.0;
    mat.sparseStatus_ = hipsparseDcsrmv( CUSPARSE,
          mat.operation_,
          mat.rows_, mat.cols_, mat.nnz_, 
          &alpha, mat.description_,
          mat.csrVal_, mat.csrRowPtr_, mat.csrColInd_,
          vec.data(), &beta,
          out.data());
    mat.checkError_("hipsparseDcsrmv() in operator*(CudaMatrix, CudaArray)");
    return CudaArray(std::move(out));
}



// Scalar multiplications with matrix:
CudaMatrix equelleCUDA::operator*(const CudaMatrix& lhs, const Scalar rhs) {
    return (rhs * lhs);
}

CudaMatrix equelleCUDA::operator*(const Scalar lhs, const CudaMatrix& rhs) {
    // rhs should not be empty
    if ( rhs.isEmpty() ) {
  OPM_THROW(std::runtime_error, "Calling CudaMatrix * Scalar with empty matrix...");
    }
    
    CudaMatrix out(rhs);
    kernelSetup s(out.nnz_);
    wrapCudaArray::scalMultColl_kernel<<<s.grid, s.block>>>(out.csrVal_,
                  lhs,
                  out.nnz_);
    return CudaMatrix(std::move(out));
}

CudaMatrix equelleCUDA::operator-(const CudaMatrix& arg) {
    return -1.0*arg;
}



// Diagonal multiplyer:
CudaMatrix CudaMatrix::diagonalMultiply(const CudaMatrix& rhs) const {
    // Make sure we do not call this function if this is not diagonal
    if ( !this->diagonal_ ) {
  OPM_THROW(std::runtime_error, "Error in CudaMatrix::diagonalMultiply\n\tCaller matrix is not diagonal!");
    }

    CudaMatrix out = rhs;
    // this is a square matrix
    kernelSetup s(this->rows_);
    wrapCudaMatrix::diagMult_kernel<<<s.grid, s.block>>>(out.csrVal_,
               out.csrRowPtr_,
               this->csrVal_,
               this->rows_);
    return CudaMatrix(std::move(out));
}

// KERNELS -------------------------------------------------


__global__ void wrapCudaMatrix::initIdentityMatrix(double* csrVal,
               int* csrRowPtr,
               int* csrColInd,
               const int nnz)
{
    const int i = myID();
    if ( i < nnz + 1) {
  csrRowPtr[i] = i;
  if (i < nnz) {
      csrVal[i] = 1.0;
      csrColInd[i] = i;
  }
    }
}


__global__ void wrapCudaMatrix::initDiagonalMatrix( double* csrVal,
                int* csrRowPtr,
                int* csrColInd,
                const double* scalars,
                const int nnz)
{
    const int i = myID();
    if ( i < nnz + 1) {
  csrRowPtr[i] = i;
  if ( i < nnz) {
      csrVal[i] = scalars[i];
      csrColInd[i] = i;
  }
    }
}


// Restriction matrix initialization kernel
__global__ void wrapCudaMatrix::initRestrictionMatrix( double* csrVal,
                   int* csrRowPtr,
                   int* csrColInd,
                   const int* set,
                   const int rows) {
    // Matrix is flat, more cols than rows.
    //   - each row has one element, hence csrRowPtr = [0,1,2,...,rows_] (size rows+1)
    //   - all nnz elements are 1, hence csrVal = [1,1,1,...,1] (size rows)
    //   - csrColInd = to_set (size rows)
    const int i = myID();
    if ( i < rows + 1) {
  csrRowPtr[i] = i;
  if ( i < rows ) {
      csrVal[i] = 1;
      csrColInd[i] = set[i];
  }
    }
}


__global__ void wrapCudaMatrix::initBooleanDiagonal( double* csrVal,
                 int* csrRowPtr,
                 int* csrColInd,
                 const bool* bool_ptr,
                 const int rows) {
    const int i = myID();
    if ( i < rows + 1) {
  csrRowPtr[i] = i;
  if ( i < rows ) {
      csrColInd[i] = i;
      if (bool_ptr[i])
    csrVal[i] = 1;
      else
    csrVal[i] = 0;
  }
    }
}


__global__ void wrapCudaMatrix::diagMult_kernel( double* csrVal,
             const int* csrRowPtr,
             const double* diagVals,
             const int total_rows) 
{
    const int row = myID();
    if ( row < total_rows ) {
  for (int i = csrRowPtr[row]; i < csrRowPtr[row+1]; i++) {
      csrVal[i] = diagVals[row] * csrVal[i];
  }
    }
}


#include "hip/hip_runtime.h"

#include <string>
#include <iostream>
#include <vector>

#include <opm/common/ErrorMacros.hpp>

#include "CollOfScalar.hpp"
#include "CudaArray.hpp"
#include "CudaMatrix.hpp"

#include <thrust/detail/raw_pointer_cast.h>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>

using namespace equelleCUDA;
using namespace wrapCudaArray;

CollOfScalar::CollOfScalar() 
    : val_(),
      der_(),
      autodiff_(false)
{
    // Intentionally left empty
}

CollOfScalar::CollOfScalar(const int size)
    : val_(size),
      der_(),
      autodiff_(false)
{
    // Intentionally left empty
}

CollOfScalar::CollOfScalar(const int size, const double value)
    : val_(size, value),
      der_(),
      autodiff_(false)
{
    // Intentionally left emtpy
}

CollOfScalar::CollOfScalar(const CudaArray& val)
    : val_(val),
      der_(),
      autodiff_(false)
{
    // Intentionally left emtpy
}

CollOfScalar::CollOfScalar(CudaArray&& val)
    : val_(std::move(val)),
      der_(),
      autodiff_(false)
{
}

CollOfScalar::CollOfScalar(const std::vector<double>& host_vec)
    : val_(host_vec),
      der_(),
      autodiff_(false)
{
    // Intentionally left emtpy
}


// Primary variable constructor
CollOfScalar::CollOfScalar(const CollOfScalar& val, const bool primaryVariable)
    : val_(val.val_),
      der_(val.size()),
      autodiff_(true)
{
    // It makes no sence to use this constructor with primaryVariable = false,
    // so we check that it is used correctly:
    if ( !primaryVariable ) {
	OPM_THROW(std::runtime_error, "Trying to create a primary variable with primaryVarible = " << primaryVariable );
    }
}

// Constructor from CudaArray and CudaMatrix
CollOfScalar::CollOfScalar(const CudaArray& val, const CudaMatrix& der)
    : val_(val),
      der_(der),
      autodiff_(true)
{
    // Intentionally left empty
}

// Move constructor from CudaArray and CudaMatrix.
// Both val and der are moved.
CollOfScalar::CollOfScalar(CudaArray&& val, CudaMatrix&& der) noexcept
    : val_(std::move(val)),
      der_(std::move(der)),
      autodiff_(true)
{
}

// Move constructor from CudaArray and CudaMatrix
// Only der is moved. val is copied.
CollOfScalar::CollOfScalar(const CudaArray& val, CudaMatrix&& der)
    : val_(val),
      der_(std::move(der)),
      autodiff_(true)
{
}


// Move constructor from CudaArray and CudaMatrix
// Only der is moved. val is copied.
CollOfScalar::CollOfScalar(CudaArray&& val, const CudaMatrix& der)
    : val_(std::move(val)),
      der_(der),
      autodiff_(true)
{
}

// Copy constructor
CollOfScalar::CollOfScalar(const CollOfScalar& coll)
    : val_(coll.val_),
      der_(coll.der_),
      autodiff_(coll.autodiff_)
{
    // Intentionally left emtpy
}

// Move constructor
CollOfScalar::CollOfScalar(CollOfScalar&& coll)
    : val_(std::move(coll.val_)),
      der_(std::move(coll.der_)),
      autodiff_(coll.autodiff_) 
{
}

// Assignment copy operator
CollOfScalar& CollOfScalar::operator= (const CollOfScalar& other)
{
    // Protect against self assignment:
    if (this != &other) {
	val_ = other.val_;
	autodiff_ = other.autodiff_;
	if ( autodiff_ ) {
	    der_ = other.der_;
	}
    }
    return *this;
}

// Move assignment operator
CollOfScalar& CollOfScalar::operator=(CollOfScalar&& other)
{
    val_ = std::move(other.val_);
    autodiff_ = other.autodiff_;
    if ( autodiff_ ) {
        der_ = std::move(other.der_);
    }
    return *this;
}

// Move compound operator for multiplication
CollOfScalar& CollOfScalar::operator*=(const Scalar lhs)
{
    val_ *= lhs;
    if ( autodiff_ ) {
        der_ *= lhs;
    }
    return *this;
}

CollOfScalar& CollOfScalar::operator*=(const CollOfScalar& rhs) {
    if ( autodiff_ || rhs.autodiff_ ) {
        // (u*v)' = u'*v + v'*u = diag(v)*u' + diag(u)*v'
        // where u = lhs and v = rhs

        der_ = der_.diagonalMultiply(rhs.val_) + rhs.der_.diagonalMultiply(val_);
        autodiff_ = true;
    }
    val_ *= rhs.val_;
    return *this;
}

CollOfScalar::~CollOfScalar()
{
    // Intentionally left blank as val_ knows how to delete itself.
}

// Member functions that only have to return val_'s function:
const double* CollOfScalar::data() const {
    return val_.data();
}

double* CollOfScalar::data() {
    return val_.data();
}

bool CollOfScalar::useAutoDiff() const {
    return autodiff_;
}

kernelSetup CollOfScalar::setup() const {
    return val_.setup();
}

std::vector<double> CollOfScalar::copyToHost() const {
    return val_.copyToHost();
}

hostMat CollOfScalar::matrixToHost() const {
    if ( !autodiff_ ) {
	OPM_THROW(std::runtime_error, "Trying to copy empty matrix to host\n");
    }
    return der_.toHost();
}

int CollOfScalar::size() const {
    return val_.size();
}

CudaMatrix CollOfScalar::derivative() const {
    return der_;
}

CudaArray CollOfScalar::value() const {
    return val_;
}

// Reduction
double CollOfScalar::reduce(const EquelleReduce reduce) const {
    // Copy to a device vector?
    thrust::device_vector<double> vec(this->size());
    double* vec_ptr = thrust::raw_pointer_cast( &vec[0] );

    hipError_t stat = hipMemcpy( vec_ptr, this->data(), 
				   this->size()*sizeof(double), 
				   hipMemcpyDeviceToDevice);
    if ( stat != hipSuccess ) {
	OPM_THROW(std::runtime_error, "Error in hipMemcpy in CollOfScalar::reduce(const EquelleReduce) with EquelleReduce = " << reduce);
    }
    
    double result = 0;
    
    //thrust::iterator<double>
    //const thrust::device_ptr<double> start(x.data());
    //const thrust::device_ptr<double> end(x.data() + x.size());
    if ( reduce == SUM ) {
	result = thrust::reduce(vec.begin(), vec.end(), 
				(double) 0, thrust::plus<double>());
    }
    else if ( reduce == PRODUCT ) {
	result = thrust::reduce(vec.begin(), vec.end(),
				(double) 1.0, thrust::multiplies<double>());
    }
    else if ( reduce == MAX ) {
	double init = -1.0*std::numeric_limits<double>::max();
	result = thrust::reduce( vec.begin(), vec.end(), init, thrust::maximum<double>());
    }
    else if ( reduce == MIN ) {
	double init = std::numeric_limits<double>::max();
	result = thrust::reduce( vec.begin(), vec.end(), init, thrust::minimum<double>());
    }
    return result;
} // reduce

// Get referance to the CudaArray with the values:
//const CudaArray& CollOfScalar::val() const {
//    return val_;
//}




// ------- ARITHMETIC OPERATIONS --------------------

CollOfScalar equelleCUDA::operator+ (const CollOfScalar& lhs,
				     const CollOfScalar& rhs)
{
    //CudaArray val = lhs.val_ + rhs.val_;
    CudaArray val = lhs.val_ + rhs.val_;
    if (lhs.autodiff_ || rhs.autodiff_) {
	CudaMatrix der = lhs.der_ + rhs.der_;
	return CollOfScalar(std::move(val), std::move(der));
    }
    return CollOfScalar(std::move(val));
}

CollOfScalar equelleCUDA::operator-(const CollOfScalar& lhs, const CollOfScalar& rhs) {
    CudaArray val = lhs.val_ - rhs.val_;
    if ( lhs.autodiff_ || rhs.autodiff_ ) {
	CudaMatrix der = lhs.der_ - rhs.der_;
	return CollOfScalar(std::move(val), std::move(der));
    }
    return CollOfScalar(std::move(val));
}

//CollOfScalar equelleCUDA::operator-(const CollOfScalar& lhs, const CollOfScalar& rhs) {
//    CollOfScalar out;
//    out.val_ = lhs.val_ - rhs.val_;
//    if ( lhs.autodiff_ || rhs.autodiff_ ) {
//	out.autodiff_ = true;
//	out.der_ = lhs.der_ - rhs.der_;
//    }
//    return out;
//}


CollOfScalar equelleCUDA::operator*(const CollOfScalar& lhs, const CollOfScalar& rhs) {
    CudaArray val = lhs.val_ * rhs.val_;
    if ( lhs.autodiff_ || rhs.autodiff_ ) {
	// (u*v)' = u'*v + v'*u = diag(v)*u' + diag(u)*v'
	// where u = lhs and v = rhs
	CudaMatrix diag_u(lhs.val_);
	CudaMatrix diag_v(rhs.val_);
	CudaMatrix der = diag_v*lhs.der_ + diag_u*rhs.der_;
	return CollOfScalar(std::move(val), std::move(der));
    }
    return CollOfScalar(std::move(val));
}

CollOfScalar equelleCUDA::operator*(CollOfScalar&& lhs, CollOfScalar&& rhs) {
    if (lhs.autodiff_) {
        lhs *= rhs;
        return CollOfScalar(std::move(lhs));
    }
    else {
        rhs *= lhs;
        return CollOfScalar(std::move(rhs));
    }
}

CollOfScalar equelleCUDA::operator/(const CollOfScalar& lhs, const CollOfScalar& rhs) {
    CudaArray val = lhs.val_ / rhs.val_;
    if ( lhs.autodiff_ || rhs.autodiff_ ) {
	// (u/v)' = (u'*v - v'*u)/(v^2)
	// where u = lhs and v = rhs
	CudaMatrix diag_u(lhs.val_); // D1
	CudaMatrix diag_v(rhs.val_); // D2
	CudaMatrix inv_v_squared( 1.0/(rhs.val_ * rhs.val_));
	CudaMatrix der = inv_v_squared*( diag_v*lhs.der_ - diag_u*rhs.der_);
	return CollOfScalar(std::move(val), std::move(der));
    }
    return CollOfScalar(std::move(val));
}

CollOfScalar equelleCUDA::operator/(CollOfScalar&& lhs, CollOfScalar&& rhs) {
    if ( lhs.autodiff_ || rhs.autodiff_ ) {
        // (u/v)' = (u'*v - v'*u)/(v^2)
        // where u = lhs and v = rhs
        CudaMatrix diag_u(lhs.val_); // D1
        CudaMatrix diag_v(rhs.val_); // D2
        CudaMatrix inv_v_squared( 1.0/(rhs.val_ * rhs.val_));
        lhs.der_ = inv_v_squared*( diag_v*lhs.der_ - diag_u*rhs.der_);
    }
    lhs.val_ /= rhs.val_;
    return CollOfScalar(std::move(lhs));
}

CollOfScalar equelleCUDA::operator/(const Scalar lhs, CollOfScalar&& rhs) {
    if ( rhs.autodiff_ ) {
        // (a/u)' = - (a/u^2)*u'
        // where a = lhs and u = rhs
        CudaMatrix diag_u_squared(lhs/(rhs.val_ * rhs.val_));
        rhs.der_ = -diag_u_squared*rhs.der_;
    }
    rhs.val_ = lhs / std::move(rhs.val_);
    return CollOfScalar(std::move(rhs));
}

CollOfScalar equelleCUDA::operator*(const Scalar lhs, const CollOfScalar& rhs) {
    CudaArray val = lhs * rhs.val_;
    if ( rhs.autodiff_ ) {
	CudaMatrix der = lhs * rhs.der_;
	return CollOfScalar(std::move(val), std::move(der));
    }
    return CollOfScalar(std::move(val));
}

CollOfScalar equelleCUDA::operator*(const CollOfScalar& lhs, const Scalar rhs) {
    return ( rhs * lhs);
}

CollOfScalar equelleCUDA::operator*(const Scalar lhs, CollOfScalar&& rhs) {
    rhs *= lhs;
    return CollOfScalar(std::move(rhs));
}

CollOfScalar equelleCUDA::operator/(const CollOfScalar& lhs, const Scalar rhs) {
    return ( (1/rhs) * lhs);
}

CollOfScalar equelleCUDA::operator/(const Scalar lhs, const CollOfScalar& rhs) {
    CudaArray val = lhs / rhs.val_;
    if ( rhs.autodiff_ ) {
	// (a/u)' = - (a/u^2)*u'
	// where a = lhs and u = rhs
	CudaMatrix diag_u_squared(lhs/(rhs.val_ * rhs.val_));
	CudaMatrix der = - diag_u_squared*rhs.der_;
	return CollOfScalar(std::move(val), std::move(der));
    }
    return CollOfScalar(std::move(val));
}

CollOfScalar equelleCUDA::operator-(const CollOfScalar& arg) {
    return -1.0*arg;
}


//  >
CollOfBool equelleCUDA::operator>(const CollOfScalar& lhs, const CollOfScalar& rhs) {
    return ( lhs.val_ > rhs.val_ );
}

CollOfBool equelleCUDA::operator>(const CollOfScalar& lhs, const Scalar rhs) {
    return ( lhs.val_ > rhs );
}

CollOfBool equelleCUDA::operator>(const Scalar lhs, const CollOfScalar& rhs) {
    return ( lhs > rhs.val_ );
}


// <
CollOfBool equelleCUDA::operator<(const CollOfScalar& lhs, const CollOfScalar& rhs) {
    // if   a < b   then b > a
    return rhs > lhs;
}

CollOfBool equelleCUDA::operator<(const CollOfScalar& lhs, const Scalar rhs) {
    // if  a < b  then   b > a
    return rhs > lhs;
}

CollOfBool equelleCUDA::operator<(const Scalar lhs, const CollOfScalar& rhs) {
    // if   a < b   then b > a
    return rhs > lhs;
}


// >=
CollOfBool equelleCUDA::operator>=(const CollOfScalar& lhs, const CollOfScalar& rhs) {
    return ( lhs.val_ >= rhs.val_ );
}

CollOfBool equelleCUDA::operator>=(const CollOfScalar& lhs, const Scalar rhs) {
    return ( lhs.val_ >= rhs );
}

CollOfBool equelleCUDA::operator>=(const Scalar lhs, const CollOfScalar& rhs) {
    return ( lhs >= rhs.val_ );
}


// <= 
CollOfBool equelleCUDA::operator<=(const CollOfScalar& lhs, const CollOfScalar& rhs) {
    // if   a <= b   then b >= a
    return rhs >= lhs;
}

CollOfBool equelleCUDA::operator<=(const CollOfScalar& lhs, const Scalar rhs) {
    // if  a <= b  then   b >= a
    return rhs >= lhs;
}

CollOfBool equelleCUDA::operator<=(const Scalar lhs, const CollOfScalar& rhs) {
    // if   a <= b   then b >= a
    return rhs >= lhs;
}


// ==
CollOfBool equelleCUDA::operator==(const CollOfScalar& lhs, const CollOfScalar& rhs) {
    return ( lhs.val_ == rhs.val_ );
}

CollOfBool equelleCUDA::operator==(const CollOfScalar& lhs, const Scalar rhs) {
    return ( lhs.val_ == rhs );
}

CollOfBool equelleCUDA::operator==(const Scalar lhs, const CollOfScalar& rhs) {
    return (rhs == lhs);
}


// !=
CollOfBool equelleCUDA::operator!=(const CollOfScalar& lhs, const CollOfScalar& rhs) {
    return ( lhs.val_ != rhs.val_ );
}

CollOfBool equelleCUDA::operator!=(const CollOfScalar& lhs, const Scalar rhs) {
    return ( lhs.val_ != rhs );
}

CollOfBool equelleCUDA::operator!=(const Scalar lhs, const CollOfScalar& rhs) {
    return (rhs != lhs);
}



// Matrix * CollOfScalar
CollOfScalar equelleCUDA::operator*(const CudaMatrix& mat, const CollOfScalar& coll) {
    if ( coll.useAutoDiff() ) {
	return CollOfScalar( mat * coll.value(), mat * coll.derivative() );
    }
    return CollOfScalar( mat * coll.value() );
}